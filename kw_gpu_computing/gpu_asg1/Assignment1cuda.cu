﻿
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define SIZE 10
 
__global__ void test(int *a, int *b){
    int i = threadIdx.x;
    b[i] = a[i] + 1;
}
 
int main(){
    int *a, *b;
    int *d_a, *d_b; 
 
    a = (int *)malloc(SIZE*sizeof(int));
    b = (int *)malloc(SIZE*sizeof(int));

    // 아래 assignment 에 해당하는 코드를 작성하여 전체 코드를 완성하고, CPP 파일과 동일한 output 이 출력되도록 하시오.

    hipMalloc((void**)&d_a, SIZE*sizeof(int));
    hipMalloc((void**)&d_b, SIZE*sizeof(int));
    
    for (int i = 0; i<SIZE; ++i)
    {
        a[i] = i;
        b[i] = 0;
    }
    
    
    hipMemcpy(d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice);
    test <<< 1, SIZE >>>(d_a, d_b); // launch test function
    hipMemcpy(b, d_b, SIZE*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<SIZE; i++)
        printf("b[%d] = %d\n", i, b[i]);    // print the results
 
    free(a);    // free the host memory spaces
    free(b);    // free the host memory spaces
    
    hipFree(d_a);    // free the device memory spaces 
    hipFree(d_b);    // free the device memory spaces 
    return 0;
}