#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <iostream>

#define N 10;
__global__ void add(int *a, int *b, int *c)
{
    int tid = threadIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(void)
{
    int a[N], b[N], C[N];
    int *dev_a, *dev_b, *dev_c;

    //Device 메모리 할당
    HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N*sizeof(int)));

    //Host에서 배열 'a'와 'b'를 채운다.
    for (int i=0; i<N; i++){
        a[i]=i;
        b[i]=i*i;
    }

    //생성한 배열 'a'와 'b'를 Device로 복사
    HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(int), cudamMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(int), cudamMemcpyHostToDevice));

    add<<<1,N>>> (dev_a, dev_b, dev_c);

    //배열 'c'를 Device에서 다시 Host로 복사
    HANDLE_ERROR(hipMemcpy(dev_c, c, N*sizeof(int), cudamMemcpyDeviceToHost));

    //결과 출력
    for (int i=0; i<N; i++){
        printf("%d + %d = %d\n", a[i], b[i], c[i];);
    }

    //Device에 할당된 메모리 해제
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}